#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <fstream>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "KeyValue.h"

#define MAX_LINES_FILE_READ 1024
#define EMITS_PER_LINE 10
#define MAX_EMITS (MAX_LINES_FILE_READ * EMITS_PER_LINE)

__host__ void loadFile(char fname[], KeyValuePair** kvs, int* length) {
	FILE* fp = fopen(fname, "r");
	if (fp == NULL)
	    exit(EXIT_FAILURE);

	char* line = NULL;
	size_t len = 0;
	int line_num = 0;
	while ((getline(&line, &len, fp)) != -1) {
	    //printf("%s", line);
	    kvs[line_num] = new KeyValuePair(line_num, line);
	    line_num ++;
	}
	fclose(fp);
	if (line)
	    free(line);
	*length = line_num;
}

__host__ __device__ void printKeyValues(KeyValuePair** kvs, int length) {
	for(int i = 0; i < length; i++) {
		if (kvs[i] == NULL) {
			printf("[%i = null]\n", i);
		} else {
			printf("%s \t %s\n", kvs[i]->key, kvs[i]->value);
		}
	}
}

__host__ __device__ void emit(KeyValuePair kv, KeyValuePair** out, int n) {
	out[n] = new KeyValuePair(kv);
}

__host__ __device__ void map(KeyValuePair kv, KeyValuePair** out, int n) {
	char* tokens = strtok(kv.value, " ,.-\t");
	int i = 0;
	while (tokens != NULL) {
		if (i >= EMITS_PER_LINE) {
			printf("WARN: Exceeded emit limit\n");
			return;
		}
		emit(KeyValuePair(tokens, "1"), out, n + i);
		tokens = strtok(NULL, " ,.-\t");
		i++;
	}
}

__host__ void cpuMap(KeyValuePair** in, KeyValuePair** out, int length) {
	for (int i = 0; i < length; i++) {
		map(*in[i], out, i * EMITS_PER_LINE);
	}
}

__global__ void kernMap(KeyValuePair** in, KeyValuePair** out, int length) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= length) return;

	map(*in[i], out, i * EMITS_PER_LINE);
}

__host__ void reduce(int start, int end, KeyValuePair** in, KeyValuePair** out, int n) {
	char* key = in[start]->key;
	char value[50];
	sprintf(value, "%i", end-start);
	out[n] = new KeyValuePair(key, value);
}

__host__ void cpuReduce(KeyValuePair** in, KeyValuePair** out, int length) {
	if (in[0] == NULL) return;

	char* key = in[0]->key;
	int start = 0;
	int n = 0;
	for (int i = 0; i < length; i++) {
		if (in[i] == NULL || strcmp(key, in[i]->key) != 0) {
			reduce(start, i, in, out, n);
			if(in[i] == NULL) {
				return; //Sorted, so we must be at the end
			}

			key = in[i]->key;
			start = i;
			n++; //TODO this math doesn't work out, ensure we can't overflow keys
		}
	}
}

__host__ int main(int argc, char* argv[]) {
	std::cout << "Running\n";
	// Load file
	int length = 0;
	KeyValuePair* file_kvs[MAX_LINES_FILE_READ] = {NULL};
	loadFile("LICENSE", file_kvs, &length);
	//printf("Length: %i\n", length);
	//printKeyValues(kvs, length);

	// Map stage
	KeyValuePair* map_kvs[MAX_EMITS] = {NULL};
	cpuMap(file_kvs, map_kvs, length);
	//printKeyValues(map_kvs, MAX_EMITS);

	//Remove any null references (stream compaction)
	//TODO

	// Sort filtered map output
	/*
	KeyValuePair** dev_map_kvs;
	int sz = MAX_EMITS * sizeof(KeyValuePair*);
	hipMalloc(&dev_map_kvs, sz);
	hipMemcpy(dev_map_kvs, map_kvs, sz, hipMemcpyHostToDevice);
	thrust::device_ptr<KeyValuePair> dev_ptr(*dev_map_kvs);
	thrust::sort(dev_ptr, dev_ptr + MAX_EMITS, KVComparator());
	*/
	std::sort(map_kvs, map_kvs + MAX_EMITS, KVComparator());
	//printKeyValues(map_kvs, MAX_EMITS);

	// Reduce stage
	KeyValuePair* reduce_kvs[MAX_EMITS] = {NULL};
	cpuReduce(map_kvs, reduce_kvs, MAX_EMITS);
	std::sort(reduce_kvs, reduce_kvs + MAX_EMITS, KVComparator());
	printKeyValues(reduce_kvs, MAX_EMITS);

	std::cout << "Done\n";
	return 0;
}
