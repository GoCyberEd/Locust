#include <string.h>
#include <stdio.h>
#include <stdlib.h>  
#include <hip/hip_runtime.h>

#include "KeyValue.h"

KeyValuePair::KeyValuePair() {
	key = NULL;
	value = NULL;
	is_empty = 1;
}

KeyValuePair::KeyValuePair(int k_num, char* v) {
	char str[100];
	sprintf(str, "%i", k_num);
	set(str, v);
}

KeyValuePair::KeyValuePair(char* k, char* v) {
	set(k, v);
}

void KeyValuePair::set(char* k, char* v) {
	char* key_ptr = (char*) malloc(sizeof(char) * strlen(k) + 1);
	strcpy(key_ptr, k);
	char* val_ptr = (char*) malloc(sizeof(char) * strlen(v) + 1);
	strcpy(val_ptr, v);

	key = key_ptr;
	value = val_ptr;
	is_empty = 0;
}

KeyValuePair* KeyValuePair::to_device() {
	char* dev_k = NULL;
	hipMalloc(&dev_k, sizeof(char) * strlen(key));
	hipMemcpy(dev_k, key, sizeof(char) * strlen(key), hipMemcpyHostToDevice);
	char *dev_v = NULL;
	hipMalloc(&dev_v, sizeof(char) * strlen(value));
	hipMemcpy(dev_v, value, sizeof(char) * strlen(value), hipMemcpyHostToDevice);

	KeyValuePair* dev_kv = NULL;
	hipMalloc((void**)&dev_kv, sizeof(KeyValuePair));
	KeyValuePair tmp_kv = KeyValuePair();
	tmp_kv.key = dev_k;
	tmp_kv.value = dev_v;
	tmp_kv.is_empty = is_empty;
	hipMemcpy(dev_kv, &tmp_kv, sizeof(KeyValuePair), hipMemcpyHostToDevice);

	return dev_kv;
}

void KeyValuePair::to_string(const KeyValuePair* kv, char* s) {
	sprintf(s, "Key: %s | Value: %s", kv->key, kv->value);
}

bool KVComparator::operator() (const KeyValuePair *kv1, const KeyValuePair *kv2) {
	if (!kv1 || !kv1->key) {
		return false;
	} else if (!kv2 || !kv2->key) {
		return true;
	}

	int i = 0;
	while(1) {
		if (kv1->key[i] != kv2->key[i]) {
			return kv1->key[i] < kv2->key[i];
		}
		i++;
	}
}
